// This program computes a simple version of matrix multiplication
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>
#include <chrono>

using std::cout;
using std::generate;
using std::vector;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void matrixMul(const int *a, const int *b, int *c, size_t N) {
  // Compute each thread's global row and column index
  size_t row = blockIdx.y * blockDim.y + threadIdx.y;
  size_t col = blockIdx.x * blockDim.x + threadIdx.x;

  // Iterate over row, and down column
  c[row * N + col] = 0;
  for (size_t k = 0; k < N; k++) {
    // Accumulate results for a single element
    c[row * N + col] += a[row * N + k] * b[k * N + col];
  }
}

// Check result on the CPU
void verify_result(vector<int> &a, vector<int> &b, vector<int> &c, int N) {
  // For every row...
  for (int i = 0; i < N; i++) {
    // For every column...
    for (int j = 0; j < N; j++) {
      // For every element in the row-column pair
      int tmp = 0;
      for (int k = 0; k < N; k++) {
        // Accumulate the partial results
        tmp += a[i * N + k] * b[k * N + j];
      }

      // Check against the CPU result
      assert(tmp == c[i * N + j]);
    }
  }
}

int main() {
  // Matrix size of 1024 x 1024;
  size_t N = 1 << 16;

  // Size (in bytes) of matrix
  size_t bytes = N * N * sizeof(int);

  // Host vectors
  vector<int> h_a(N * N);
  vector<int> h_b(N * N);
  vector<int> h_c(N * N);

  // Initialize matrices
  generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
  generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

  // Allocate device memory
  int *d_a, *d_b, *d_c;
  gpuErrchk(hipMalloc(&d_a, bytes));
  gpuErrchk(hipMalloc(&d_b, bytes));
  gpuErrchk(hipMalloc(&d_c, bytes));

  // Copy data to the device
  hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

  // Threads per CTA dimension
  int THREADS = 32;

  // Blocks per grid dimension (assumes THREADS divides N evenly)
  int BLOCKS = N / THREADS;

  // Use dim3 structs for block  and grid dimensions
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  std::cout << "Kernel starting:\n";
  auto start = std::chrono::steady_clock::now();

  // Launch kernel
  matrixMul<<<blocks, threads>>>(d_a, d_b, d_c, N);

  // Copy back to the host
  gpuErrchk(hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost));
  
  auto end = std::chrono::steady_clock::now();
  std::chrono::duration<double> time = end - start;
  cout << "Kernel Duration: " << time.count() << "\n";

  // Check result
  //verify_result(h_a, h_b, h_c, N);

  cout << "COMPLETED SUCCESSFULLY\n";

  // Free memory on device
  gpuErrchk(hipFree(d_a));
  gpuErrchk(hipFree(d_b));
  gpuErrchk(hipFree(d_c));

  return 0;
}